//compile with
//nvcc delta.cu -I /home/baggio/NVIDIA_CUDA_SDK/common/inc/ -L /home/baggio/NVIDIA_CUDA_SDK/lib/ -lcuda -lcudart -lcutil -lGL -lGLU
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

// includes, project
#include <cutil.h>
#include <cuda_gl_interop.h>
//#define TIMER
#define FINETUNING
// includes, kernels
#include <delta_kernel.cu>
#define NUMTHREADS 128





float* pixels = NULL;
float* dEdges[4];
float maxDistance = 0.0f;
int* hSource = NULL;

int GN = 262144;
int maxRCount = 0;

hipArray* array[4];


struct edge{
  int dNode[4];
  float weight[4];
};

edge* nodes;





void
runTest( int argc, char** argv,int iw, int ih, int startNode);

void 
runTestGraph ( int argc, char** argv, int iw, int ih, int startNode, float* gradient);


void printPath(int* source, int destination){
	printf("%d <- %d\n",destination,source[destination]);
	if(source[destination]==destination) return;
	printPath( source, source[destination]);
	
}

void printPathSource(int destination, int* path){
	path[0] = hSource[destination];
	printf("%d <- %d\n",destination,hSource[destination]);
	if(hSource[destination]==destination) return;
	printPathSource( hSource[destination],path+1);
}

void loadTexture(int iw, int ih, float* data, hipArray* cArray, texture<float, 2, hipReadModeElementType>* myTex){
    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL(hipMallocArray(&cArray, &desc, iw, ih));
    CUDA_SAFE_CALL(hipMemcpyToArray(cArray, 0, 0, data, sizeof(float)*iw*ih, hipMemcpyHostToDevice));
    // Bind the array to the texture
    hipBindTextureToArray( *myTex, cArray, desc);

}

void loadGraphEdges(int iw, int ih,char* myFile){
  int n;
  FILE* in = fopen(myFile,"r");
  fscanf(in,"%d\n",&n);
  
  for(int i=0;i<4;i++){
    dEdges[i] = (float*) malloc(iw*ih*sizeof(float));
    for(int j=0;j<iw*ih;j++){
      dEdges[i][j]=INF;
    }
  }

  while(1){
    int source, dest;
    double eWeight;
    fscanf(in,"%d",&source);
    //printf("Reading %d\n",source);
    if(source==-1) break;
    fscanf(in,"%d%lf\n",&dest,&eWeight);


    //links from 512.graph are either right or down
    if(dest == source + 1){ //it's a RIGHT link
      dEdges[RIGHT][source] = eWeight;
      dEdges[LEFT] [dest  ] = eWeight;
    }
    else if(dest == source + iw){ //it's a DOWN link
      dEdges[DOWN][source] = eWeight;
      dEdges[UP]  [dest  ] = eWeight;
    }        

  }

    loadTexture(iw,ih,dEdges[0],array[0],&mytex0);
    loadTexture(iw,ih,dEdges[1],array[1],&mytex1);
    loadTexture(iw,ih,dEdges[2],array[2],&mytex2);
    loadTexture(iw,ih,dEdges[3],array[3],&mytex3);

  
}

void printDistances(float* hDist,int iw, int ih){
   for(int i=0;i<iw*ih;i++){
      if(i%iw==0) printf("%d ",i/iw);
      if(hDist[i]<INF){
        printf("%5.1f ",hDist[i]);
	if(hDist[i]>maxDistance) maxDistance=hDist[i];
      }
      else
        printf("INFINI ",hDist[i]);
      if(i%iw==iw-1) printf("\n");
    }      
    return;
}



void
runTest( int argc, char** argv, int iw, int ih, int startNode) 
{



    //initialize the device
    hipSetDevice(0);

    GN = iw*ih;


    unsigned int num_threads = NUMTHREADS;

    // setup execution parameters
    dim3  grid( 1, 1, 1);
    dim3  threads( num_threads, 1, 1);

    nodes = (edge*) malloc(GN*sizeof(edge));   

    //loadGraphEdges(iw,ih,argv[1]);


    float* dDist;
    hipMalloc( (void**) &dDist, GN*sizeof(float));
    float* hDist = (float*) malloc(GN*sizeof(float));
    
    for(int i=0;i<GN;i++){
      hDist[i]=INF;
    }
    int* dSource;
    hipMalloc( (void**) &dSource, GN*sizeof(int));
    hSource = (int*) malloc(GN*sizeof(float));
    for(int i=0;i<GN;i++){
      hSource[i]=-1;
    }



    
    int* dBucketMap;
    hipMalloc( (void**) &dBucketMap, GN*sizeof(int));
    int* hBucketMap = (int*) malloc(GN*sizeof(int));
    
     
    for(int i=0;i<GN;i++){
      hBucketMap[i]=-1;
    }

    hipMemcpy( dBucketMap, hBucketMap, GN*sizeof(int), hipMemcpyHostToDevice);

    int* dBucketPos;
    hipMalloc( (void**) &dBucketPos, GN*sizeof(int));
    int* hBucketPos = (int*) malloc(GN*sizeof(int));

    int* dB;
    hipMalloc( (void**) &dB, BUCKETSIZE*NUMBUCKETS*sizeof(int));
    int* hB = (int*) malloc(BUCKETSIZE*NUMBUCKETS*sizeof(int));
  

    printf("Bucket size %d\n",BUCKETSIZE);

    int* hBi = hB;
//    int hBiCount[1];
//    hBiCount[0] = 1;
    for(int i=0;i<BUCKETSIZE*NUMBUCKETS;i++)
      hBi[i]=-1;
    
    //start node

    hBi[0] = startNode;
    hDist[startNode] = 0.0f;

    //set startNode source node in the path as the node itself, so that the getPath function works
    hipMemcpy( dSource+startNode, &startNode, 1*sizeof(int), hipMemcpyHostToDevice);
    
    



    // copy host memory to device
    hipMemcpy( dDist, hDist, GN*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( dB, hBi, BUCKETSIZE*NUMBUCKETS*sizeof(int), hipMemcpyHostToDevice);


    int* BCount;
    hipMalloc( (void**) &BCount, NUMBUCKETS * sizeof(int));
    int* hBCount;
    hBCount = (int*) malloc(NUMBUCKETS*sizeof(int));
    for(int i=0;i<NUMBUCKETS;i++)
      hBCount[i]=0;

    hBCount[0]=1;

    // copy host memory to device
    hipMemcpy( BCount, hBCount, NUMBUCKETS*sizeof(int), hipMemcpyHostToDevice);


    int* dBPos;
    hipMalloc( (void**) &dBPos, NUMBUCKETS * sizeof(int));
    int* hBPos;
    hBPos = (int*) malloc(NUMBUCKETS*sizeof(int));
    for(int i=0;i<NUMBUCKETS;i++)
      hBPos[i]=0;
    hBPos[0]=1;
    hipMemcpy( dBPos, hBPos, NUMBUCKETS*sizeof(int), hipMemcpyHostToDevice);




    // allocate device memory for result
    int * dRLoc;
    hipMalloc( (void**) &dRLoc, GN*sizeof(int));
    int * dR;
    hipMalloc( (void**) &dR, 16*BUCKETSIZE*sizeof(int)); //R is 4 times bigger than RLoc, because UP,DOWN,LEFT and RIGHT will each have a pos in R

    int* hR = (int*) malloc(16*BUCKETSIZE*sizeof(int));
    int* hRLoc = (int*) malloc(GN*sizeof(int));
    float* hDistR= (float*) malloc(16*BUCKETSIZE*sizeof(float));


    float* dDistR;
    hipMalloc( (void**) &dDistR  , 16*BUCKETSIZE*sizeof(float));
    int* dSourceR;
    hipMalloc( (void**) &dSourceR, 16*BUCKETSIZE*sizeof(int  ));


    int * dS;
    hipMalloc( (void**) &dS, 32*BUCKETSIZE*sizeof(int));
    int * hS= (int*) malloc(32*BUCKETSIZE*sizeof(int));
    int * dSCount;
    hipMalloc( (void**) &dSCount, 1*sizeof(int));
    int * hSCount= (int*) malloc(1*sizeof(int));

    printf("Starting timer\n");

  unsigned int nvtimer = 0;
    cutCreateTimer( &nvtimer);
    cutStartTimer( nvtimer);


  unsigned int laptimer = 1;
    cutCreateTimer( &laptimer);
  float ktime;
   
  unsigned int labelTimer =2;
    cutCreateTimer( &labelTimer);
  int labelCounter=0;

  unsigned int copyTimer =3;
    cutCreateTimer( &copyTimer);
  int copyCounter=0;
    
  unsigned int relaxTimer =4;
    cutCreateTimer( &relaxTimer);
  int relaxCounter=0;


  unsigned int emptyTimer =5;
    cutCreateTimer( &emptyTimer);
  int emptyCounter=0;

  

       float* lido;
    hipMalloc( (void**) &lido, GN*sizeof(float));
     float* Hlido = (float*) malloc(GN*sizeof(float)) ;
    

      int* RCount;
      RCount = (int*) malloc(1*sizeof(int));
  
    for(int i=0;i<470;i++){
      printf("Bucket %d\n",i);
#ifdef TIMER
      printf("%d\n",i);
#endif

      hipMemcpy( RCount,  &dBPos[i],  1*sizeof(int),       hipMemcpyDeviceToHost) ;    
   

/*	printf("Before, Rcount[0] was %d\n",RCount[0]);
      for(int j=0;j<30;j++){
	printf("Bcount[%d]=%d\n",j,debugVector[j]);
      } */


      //S <- EMPTY
      hSCount[0]=0;
      hipMemcpy( dSCount, hSCount,1*sizeof(int), hipMemcpyHostToDevice);
    
//  printf("Before RCount %d(i=%d)\n",RCount[0],i);
  //    printf("Still safe%d\n",i);

      //While B[i] != EMPTY
      int sameCount = 0;
      cutStartTimer( laptimer);
      while(RCount[0]!=0){
        if(RCount[0]>maxRCount) maxRCount = RCount[0];
	printf("RCount %d\n",RCount[0]);
//      if(RCount[0]==0) i++;
        sameCount++;

#ifdef TIMER
        cutStartTimer( laptimer);
#endif

#ifdef FINETUNING
        cutStartTimer( labelTimer);
#endif
	
        labelKernel    <<<grid, threads >>> ( i, dB, BCount,dBPos, iw,ih, dRLoc, dR, dDistR, dSourceR, dDist,dBucketMap);
        hipDeviceSynchronize();
    


#ifdef FINETUNING
        cutStopTimer( labelTimer);
	labelCounter++;
#endif

        hipMemcpy( &hBPos[i],  &dBPos[i],  1*sizeof(int),       hipMemcpyDeviceToHost) ;


#ifdef TIMER
        ktime = cutGetTimerValue( laptimer );
        printf("Label         kernel %f\n",ktime);	
        cutStartTimer( laptimer);
#endif

#ifdef FINETUNING
        cutStartTimer( copyTimer);
#endif



        copyB2SKernel  <<<grid, threads >>> ( i, dB, BCount,dBPos, dS, dSCount);
        hipDeviceSynchronize();

#ifdef FINETUNING
        cutStopTimer( copyTimer);
	copyCounter++;
#endif



#ifdef TIMER
        ktime = cutGetTimerValue( laptimer );
        printf("CopyB2S       kernel %f\n",ktime);	
#endif

#ifdef TIMER
        cutStartTimer( laptimer);
#endif


#ifdef FINETUNING
        cutStartTimer( emptyTimer);
#endif


//        emptyKernel    <<<grid, threads >>> ();
//        hipDeviceSynchronize();

#ifdef FINETUNING
        cutStopTimer( emptyTimer);
	emptyCounter++;
#endif


#ifdef TIMER
        ktime = cutGetTimerValue ( laptimer );
	printf(" Empty        kernel %f\n",ktime);
#endif

        //todo: correct RCount
        cutStartTimer( laptimer);

	
//        printf("Sending RCount %d\n",4*4*hBPos[i]);

#ifdef FINETUNING
        cutStartTimer( relaxTimer);
#endif

        relaxKernelPath    <<<grid, threads >>> ( 4*4*hBPos[i], dB, BCount, dBPos, dRLoc, dR, dDistR, dDist,dBucketPos, dBucketMap,lido, dSourceR, dSource);
        hipDeviceSynchronize();

#ifdef FINETUNING
        cutStopTimer( relaxTimer);
	relaxCounter++;
#endif
 

#ifdef TIMER
        ktime = cutGetTimerValue( laptimer );
        printf("Relaxing      kernel %f\n",ktime);
#endif


        hipMemcpy( RCount,  &BCount[i],  1*sizeof(int),       hipMemcpyDeviceToHost) ;
        hipMemcpy( hSCount,  dSCount,  1*sizeof(int),       hipMemcpyDeviceToHost) ;        
	

#ifdef TIMER
        printf("After RCount %d (i=%d) | SCount %d\n",RCount[0],i,hSCount[0]);
#endif


      }

//      ktime = cutGetTimerValue( laptimer );
#ifdef TIMER
      printf("Same called %d times.\n",sameCount);
#endif

//      printf("Label kernel %f\n",ktime);	
// 	printf("Before heavy label%d\n",i);      


//      cutStartTimer( laptimerrunTestGraph);

//      labelHeavyKernel    <<<grid, threads >>> ( 0, dS, hSCount[0], iw,ih, dRLoc, dR, dDistR, dDist,dBucketMap);
//      hipDeviceSynchronize();



      ktime = cutGetTimerValue( laptimer );
#ifdef TIMER
      printf("Labelling heavy kernel %f\n",ktime);
#endif

// 	printf("After heavy label%d\n",i);      
      //todo: correct SCount

#ifdef TIMER
      cutStartTimer( laptimer);
#endif
//      int temp[1];
//      hipMemcpy( temp,  &dSCount[0],  1*sizeof(int),       hipMemcpyDeviceToHost) ;
//      relaxKernel         <<<grid, threads >>> ( temp[0], dB, BCount, dRLoc, dR, dDistR, dDist,dBucketPos, dBucketMap,lido);

//      relaxKernel    <<<grid, threads >>> ( 4*4*hSCount[0], dB, BCount, dBPos, dRLoc, dR, dDistR, dDist,dBucketPos, dBucketMap,lido);

//      ktime = cutGetTimerValue( laptimer );
#ifdef TIMER
      printf("Relaxing heavy  kernel %f\n",ktime);
#endif

// 	printf("After heavy relax%d\n",i);      

//      printf("Done here. i = %d\n",i);


    }


    // check if kernel execution generated and error

   CUT_CHECK_ERROR("Kernel execution failed");

 

    ktime = cutGetTimerValue( nvtimer );
    hipMemcpy( hRLoc,  dRLoc,  1024*sizeof(int),       hipMemcpyDeviceToHost) ;    
    hipMemcpy( hRLoc,  dRLoc,  1024*sizeof(int),       hipMemcpyDeviceToHost) ;
    hipMemcpy( hR,     dR,     4*1024*8*sizeof(int),   hipMemcpyDeviceToHost) ;
    hipMemcpy( hDistR, dDistR, 4*1024*8*sizeof(float), hipMemcpyDeviceToHost) ;
    hipMemcpy( hS,     dS,     1024*8*sizeof(float),   hipMemcpyDeviceToHost) ;
    hipMemcpy( hDist,  dDist,  GN*sizeof(int),       hipMemcpyDeviceToHost) ;
    hipMemcpy( hBCount, BCount,  NUMBUCKETS*sizeof(int),       hipMemcpyDeviceToHost) ;
    hipMemcpy( hB, dB,           2*NUMBUCKETS*sizeof(int),       hipMemcpyDeviceToHost) ;

    hipMemcpy( Hlido, lido, GN*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy( hR, dR, GN*sizeof(int),hipMemcpyDeviceToHost);

    hipMemcpy( hSource, dSource, GN*sizeof(int),  hipMemcpyDeviceToHost);

    

  printDistances(hDist,iw,ih); 

    printPath(hSource, 10);

#ifdef FINETUNING
    printf("Label averageTime: %f ms (called %d times)\n",cutGetAverageTimerValue(labelTimer), labelCounter);
    printf("Copy  averageTime: %f ms (called %d times)\n",cutGetAverageTimerValue(copyTimer),  copyCounter);
    printf("Relax averageTime: %f ms (called %d times)\n",cutGetAverageTimerValue(relaxTimer), relaxCounter);
    printf("Empty averageTime: %f ms (called %d times)\n",cutGetAverageTimerValue(emptyTimer), emptyCounter);
    printf("Num threads: %d\n",num_threads);
    printf("Max RCount %d\n",maxRCount);
    printf("Max Distance %f\n", maxDistance);
#endif 

    for(int i=0;i<5;i++){
      printf("source[%d]= %d\n",i,hSource[i]);
    }
    for(int i=512;i<517;i++){
      printf("source[%d]= %d\n",i,hSource[i]);
    }

    printf("It took: %f ms\n", ktime);

    hipUnbindTexture(mytex0);
    hipUnbindTexture(mytex1);
    hipUnbindTexture(mytex2);
    hipUnbindTexture(mytex3);

    hipFreeArray(array[0]);
    hipFree(dB);
    hipFree(dRLoc);
    hipFree(dDist);
    hipFree(dBucketMap);
    hipFree(dBucketPos);

    hipFree(BCount);

    hipFree(dR);
    hipFree(dDistR);
    hipFree(dSource);
    hipFree(dS);
    hipFree(dSCount);
    free(dEdges[0]);
    free(dEdges[1]);
    free(dEdges[2]);
    free(dEdges[3]);
    free(nodes);
    
 
}

void empty(){
	dim3  grid( 1, 1, 1);
    dim3  threads( 256, 1, 1);
    emptyKernel    <<<grid, threads >>> ();
}

void loadEdgesFromGradient(int iw, int ih, float* gradient){

  for(int i=0;i<4;i++){
    dEdges[i] = (float*) malloc(iw*ih*sizeof(float));
    for(int j=0;j<iw*ih;j++){
      dEdges[i][j]=INF;
    }
  }

  for(int i=1;i<ih;i++){
  	for(int j=0;j<iw;j++){
  		int index = i*iw+j;
  		dEdges[RIGHT][index]    = 0.5*(1 - gradient[index]);
  		dEdges[LEFT ][index+1]  = 0.5*(1 - gradient[index+1]);
  		dEdges[DOWN] [index]    = 0.5*(1 - gradient[index]);
  		dEdges[UP]   [index-iw] = 0.5*(1 - gradient[index-iw]);
  		//printf("up %f\n",dEdges[RIGHT][index]); 
  	}
  	
  }  
    loadTexture(iw,ih,dEdges[0],array[0],&mytex0);
    loadTexture(iw,ih,dEdges[1],array[1],&mytex1);
    loadTexture(iw,ih,dEdges[2],array[2],&mytex2);
    loadTexture(iw,ih,dEdges[3],array[3],&mytex3);

}

void 
runTestGraph ( int argc, char** argv, int iw, int ih, int startNode, float* gradient){
	//loading edges from gradient
	loadEdgesFromGradient(iw,ih,gradient);	
	printf("Gradient %f\n",gradient[startNode]);
	runTest(argc,argv,iw,ih,startNode);
	printf("Delta for %d finished\n",startNode);
	
	
}

