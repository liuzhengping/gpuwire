#include "hip/hip_runtime.h"
#include <stdio.h>
__global__ void
emptyKernel(){
}



extern "C" void 
call(){
    dim3  grid( 1, 1, 1);
    dim3  threads( 256, 1, 1);
	emptyKernel    <<<grid, threads >>> ();
	hipDeviceSynchronize();
	printf("Called\n");
}

