#include "hip/hip_runtime.h"
#ifndef _DELTA_KERNEL_H_
#define _DELTA_KERNEL_H_

#include <stdio.h>
#define DELTA 1.0
#define INF 1e20
#define BUCKETSIZE 4096*8*2
#define NUMBUCKETS 512

#define DOWN  0
#define UP    1
#define RIGHT 2
#define LEFT  3


//#define EMULATION

#ifdef EMULATION
#define DEBUG(x...) printf(x)
#else
#define DEBUG(x...) 
#endif

texture<float, 2, hipReadModeElementType> mytex0;
texture<float, 2, hipReadModeElementType> mytex1;
texture<float, 2, hipReadModeElementType> mytex2;
texture<float, 2, hipReadModeElementType> mytex3;

__device__ void
demptyKernel(){
}

__global__ void
labelKernel (int i, int* B,int* BCount,int* BPos, int tw,int th, int* RLoc,int* R, float* dR,int* dSourceR, float* d,int* vBucketMap){
//ver se não precisa colocar 4 BiCount em algum lugar...

  //todo: try to increase speed using shared memory for RLoc... think more about it (maybe RLoc is too big for shared memory)
  const unsigned int tid = threadIdx.x;
  const unsigned int num_threads = blockDim.x;

  int BiCount = BPos[i];
  int node, row, col,index;
  float cost,f1,f2,fmin;

  //cleaning R
  //4 times because each node can be reached from up, down, left and right directions (and more 4 times because for each node 4 more are open)
//  DEBUG("BiCount %d lastpos %d\n",BiCount,4*(BiCount-1)+3);
  for(int k=0;  (num_threads*k + tid) < 16*BiCount;k++){
    R[(num_threads*k + tid)  ]=-1;
    dR[(num_threads*k + tid)  ]=INF;
    dSourceR[(num_threads*k + tid)  ]=-1;
  }


  __syncthreads();
  const int dx[4]={0,0,1,-1};
  const int dy[4]={1,-1,0,0};
  

  for(int k=0; num_threads*k + tid < BiCount;k++){
    node = B  [ BUCKETSIZE*i + num_threads*k + tid];
    if(node!=-1){

      DEBUG("(tid %d) node %d(from B[%d] pos %d)\n",tid,node,i,num_threads*k + tid);
      for(int j=0;j<4;j++){
        switch(j){
	  case 0:
           cost = tex2D(mytex0,node%tw,node/tw);
           break;
	  case 1:
           cost = tex2D(mytex1,node%tw,node/tw);
           break;
	  case 2:
           cost = tex2D(mytex2,node%tw,node/tw);
           break;
	  case 3:
           cost = tex2D(mytex3,node%tw,node/tw);
           break;
        }
        row = node/tw + dy[j];
        col = node%tw + dx[j];

        if( (row>=0) && (row < th) && (col >= 0) && (col < tw) ){
          RLoc [ row*tw + col ] = 4*(num_threads*k + tid)+j;
          DEBUG("(tid %d)Connecting node %d to be processed by %d\n",tid,row*tw+col,RLoc [ row*tw + col ]);
        }

      }

    }
  }

  __syncthreads();

  //copy Edges to R
  for(int k=0; num_threads*k + tid < BiCount;k++){
    node = B  [ BUCKETSIZE*i + num_threads*k + tid];
    if(node!=-1){
      for(int j=0;j<4;j++){
        switch(j){
  	case 0:
         cost = tex2D(mytex0,node%tw,node/tw);
         break;
	case 1:
         cost = tex2D(mytex1,node%tw,node/tw);
         break;
	case 2:
         cost = tex2D(mytex2,node%tw,node/tw);
         break;
	case 3:
         cost = tex2D(mytex3,node%tw,node/tw);
         break;
      }



        row = node/tw + dy[j];
        col = node%tw + dx[j];
              DEBUG("Pre-candidate in R %d (d=%f)\n",row*tw + col,d[node]+cost,dR [4*RLoc[row*tw + col]+j]);
        if( (row>=0) && (row < th) && (col >= 0) && (col < tw) ){ 
            DEBUG("Candidate in R %d (d=%f)\n",row*tw + col,d[node]+cost,dR [4*RLoc[row*tw + col]+j]);
          if((cost<=DELTA)&&( d[node]+cost < d[row*tw + col])){
              R [4*RLoc[row*tw + col]+j] = row*tw + col;
              dR[4*RLoc[row*tw + col]+j] = d[node]+cost;
	      dSourceR[4*RLoc[row*tw + col]+j]=node;
              DEBUG("New node in R %d (d=%f,e=%f) in pos %d\n",R [4*RLoc[row*tw + col]+j],dR [4*RLoc[row*tw + col]+j],cost,4*RLoc[row*tw + col]+j);
              vBucketMap[node]=-1;
          }
        }
      }
    }
  }
  __syncthreads();

  //gathering data to find the minimum cost way to get to node n
  //TODO: OPTIMIZE IN SUCH A WAY IT WON'T BE NEEDED TO GO THROUGH THE 4 EDGES, since they store the same value
  int smin ;
  float dists[4];
  for(int k=0;  (num_threads*k + tid) < 4*BiCount;k++){
    index    = 4*(num_threads*k + tid);
    dists[0] = dR[ index  ];
    dists[1] = dR[ index+1];
    dists[2] = dR[ index+2];
    dists[3] = dR[ index+3];

    //finds the node with minimum distance, so that the path can be stored
    if( dists[0] < dists[1]){
	if( dists[2] < dists[3]){
	  if( dists[0] < dists[2]){
	    smin = dSourceR[index];
	  }
          else{
            smin = dSourceR[index+2];
	  }
	}
        else{
	  if( dists[0] < dists[3]){
	    smin = dSourceR[index];
	  }
	  else{
	    smin = dSourceR[index+3];
	  }
	}
    }
    else{
	if( dists[2] < dists[3]){
	  if( dists[1] < dists[2]){
	    smin = dSourceR[index+1];
	  }
          else{
            smin = dSourceR[index+2];
	  }
	}
        else{
	  if( dists[1] < dists[3]){
	    smin = dSourceR[index+1];
	  }
	  else{
	    smin = dSourceR[index+3];
	  }
	}

    }    

    f1 = fminf( dists[0] , dists[1] );
    f2 = fminf( dists[2] , dists[3] );

    fmin = fminf(f1,f2);
    dR[index  ]=fmin;
    dR[index+1]=fmin;
    dR[index+2]=fmin;
    dR[index+3]=fmin;

    DEBUG("Smin %d\n",smin);
    dSourceR[index  ]=smin;
    dSourceR[index+1]=smin;
    dSourceR[index+2]=smin;
    dSourceR[index+3]=smin;

    
  }
  __syncthreads();

}

//Pensar se o fato de S ter duplicatas pode impactar em algo
__global__ void 
copyB2SKernel(int i, int* B, int* BCount,int* BPos, int* S, int* SCount){
  //TODO: optimize this code
  //there's an optimized way of doing this, which is by only
  //storing SCount = Scount+ BCount, as output
  //and controlling with local variables thread positions

  const unsigned int tid = threadIdx.x;
  const unsigned int num_threads = blockDim.x;

  int pos;

  int BiCount = BCount[i];
  for(int k=0; num_threads*k + tid < BiCount;k++){
    if(B[i*BUCKETSIZE+num_threads*k+tid]!=-1){
      pos = atomicAdd(&SCount[0],1);
      S[pos] = B[i*BUCKETSIZE+num_threads*k+tid];
    }
  }
  __syncthreads();
  BCount[i]=0;
  BPos[i]=0;
  __syncthreads();
//  DEBUG("(tid %d) SCount %d\n",tid,SCount[0]);


}


//Parallel relax edges
__global__ void
relaxKernelPath( int RCount, int* B,int* BCount,int* BPos, int* RLoc,int* R,float*  dR,float* d, int* vBucketLoc, int* vBucketMap, float* deb, int* dSourceR, int* dSource){

  const unsigned int tid = threadIdx.x;
  const unsigned int num_threads = blockDim.x;
  int v,bn,bn_old, index;
  float x;


  //remove node from old bucket



  for(int k=0; num_threads*k + tid < RCount;k++){
    index = num_threads*k + tid;

    if(R[index]!=-1){

      x = dR[index];      
      v = R[index];

      if(x<d[v]){

        bn_old = vBucketMap[v];
        if (bn_old != -1) {
          int oldIndex = bn_old*BUCKETSIZE+vBucketLoc[v];
//	  
          B[oldIndex] = -1;//GN;
//          int oldc = atomicSub(&BCount[bn_old],1);
	  atomicSub(&BCount[bn_old],1);

//        printf("Removing %d from %d(%d)\n",v,bn_old,oldc);


        }
      }

    }    

  }




  __syncthreads();



  for(int k=0; num_threads*k + tid < RCount;k++){

    if(R[num_threads*k + tid]!=-1){
//deb[0]= (float) (BUCKETSIZE);
      x = dR[num_threads*k + tid];
      v = R[num_threads*k + tid];
      if(x < d[v]){

	      bn = (int) (dR[num_threads*k + tid]/DELTA);

	      atomicAdd(&BCount[bn],1);
	      int pos = atomicAdd(&BPos[bn],1);
	      DEBUG("Pos %d BCount[%d] %d node %d (x=%f)\n",pos,bn,BPos[bn],v,x);
	      
	      B[bn*BUCKETSIZE+pos] = v;
	      d[v] = x;
	      dSource[v] = dSourceR[num_threads*k+tid];
	      
	      vBucketLoc[v] = pos;
	      vBucketMap[v] = bn;
	      RLoc[v]=-1;
      }

    }
  }


  __syncthreads();

  
}



//Parallel relax edges
__global__ void
relaxKernel( int RCount, int* B,int* BCount,int* BPos, int* RLoc,int* R,float*  dR,float* d, int* vBucketLoc, int* vBucketMap, float* deb){

  const unsigned int tid = threadIdx.x;
  const unsigned int num_threads = blockDim.x;
  int v,bn,bn_old, index;
  float x;
//  int myAdd=0;
  deb[20]= (float)RCount;

  for(int k=0; num_threads*k + tid < RCount;k++){
    index = num_threads*k + tid;
    deb[2*index]= R[index];
    deb[2*index+1]= dR[index];
  }

//  DEBUG("relaxing RCount %d\n",RCount);

  //remove node from old bucket
//  RCount = RCount /4;



  for(int k=0; num_threads*k + tid < RCount;k++){
    index = num_threads*k + tid;
  //  deb[index]= R[index];

    if(R[index]!=-1){

      x = dR[index];      
      v = R[index];

      if(x<d[v]){

        bn_old = vBucketMap[v];
        if (bn_old != -1) {
          int oldIndex = bn_old*BUCKETSIZE+vBucketLoc[v];
//	  
          B[oldIndex] = -1;//GN;
//          int oldc = atomicSub(&BCount[bn_old],1);
	  atomicSub(&BCount[bn_old],1);

//        printf("Removing %d from %d(%d)\n",v,bn_old,oldc);


        }
      }

    }    

  }




  __syncthreads();






  for(int k=0; num_threads*k + tid < RCount;k++){

    if(R[num_threads*k + tid]!=-1){
//deb[0]= (float) (BUCKETSIZE);
      x = dR[num_threads*k + tid];
      v = R[num_threads*k + tid];
      if(x < d[v]){



      bn = (int) (dR[num_threads*k + tid]/DELTA); 
      
   //   printf("Bn %d\n",bn);

      atomicAdd(&BCount[bn],1);
      int pos = atomicAdd(&BPos[bn],1);
      DEBUG("Pos %d BCount[%d] %d node %d (x=%f)\n",pos,bn,BPos[bn],v,x);

      
      B[bn*BUCKETSIZE+pos] = v;
      d[v] = x;

      
      vBucketLoc[v] = pos;  
      vBucketMap[v] = bn;
      RLoc[v]=-1;


  //only debug info
//    for(int i=0;i<BPos[bn];i++){
//      DEBUG("B(%d)=%d ",i,B[bn*BUCKETSIZE+i]);
//    }
//    DEBUG("\n");



      }

    }
  }


  __syncthreads();

  
}


__global__ void
labelHeavyKernel (int i, int* B,int SCount, int tw,int th, int* RLoc,int* R, float* dR, float* d,int* vBucketMap){
  //todo: try to increase speed using shared memory for RLoc... think more about it (maybe RLoc is too big for shared memory)
  const unsigned int tid = threadIdx.x;
  const unsigned int num_threads = blockDim.x;

  int BiCount = SCount;
  int node, row, col,index;
  float cost,f1,f2,fmin;

  //cleaning R
  //4 times because each node can be reached from up, down, left and right directions (and more 4 times because for each node 4 more are open)
  //  DEBUG("BiCount %d lastpos %d\n",BiCount,4*(BiCount-1)+3);
  for(int k=0;  (num_threads*k + tid) < 16*BiCount;k++){
    index = 4*(num_threads*k + tid);
    R[index  ]=-1;
    R[index+1]=-1;
    R[index+2]=-1;
    R[index+3]=-1;
  
    dR[index  ]=INF;
    dR[index+1]=INF;
    dR[index+2]=INF;
    dR[index+3]=INF;
  }

  __syncthreads();
  const int dx[4]={0,0,1,-1};
  const int dy[4]={1,-1,0,0};  

  for(int k=0; num_threads*k + tid < BiCount;k++){
    node = B  [ BUCKETSIZE*i + num_threads*k + tid];
    if(node!=-1){

      DEBUG("(tid %d) node %d(from B[%d] pos %d)\n",tid,node,i,num_threads*k + tid);
      for(int j=0;j<4;j++){
        //TRY TO OPTIMIZE THIS PART REMOVING THE IF AND SETTING IMPOSSIBLE EDGE VALUES TO INFINITY
        switch(j){
	  case 0:
           cost = tex2D(mytex0,node%tw,node/tw);
           break;
	  case 1:
           cost = tex2D(mytex1,node%tw,node/tw);
           break;
	  case 2:
           cost = tex2D(mytex2,node%tw,node/tw);
           break;
	  case 3:
           cost = tex2D(mytex3,node%tw,node/tw);
           break;
        }
//      cost = tex2D(mytex[j],node%tw,node/tw);//todo: change texture
        row = node/tw + dy[j];
        col = node%tw + dx[j];

        if( (row>=0) && (row < th) && (col >= 0) && (col < tw) ){
          RLoc [ row*tw + col ] = 4*(num_threads*k + tid)+j;
          DEBUG("(tid %d)Connecting node %d to be processed by %d\n",tid,row*tw+col,RLoc [ row*tw + col ]);
        }

      }

    }
  }

  __syncthreads();

  //copy Edges to R
  for(int k=0; num_threads*k + tid < BiCount;k++){
    node = B  [ BUCKETSIZE*i + num_threads*k + tid];
    if(node!=-1){
      for(int j=0;j<4;j++){
        switch(j){
  	case 0:
         cost = tex2D(mytex0,node%tw,node/tw);
         break;
	case 1:
         cost = tex2D(mytex1,node%tw,node/tw);
         break;
	case 2:
         cost = tex2D(mytex2,node%tw,node/tw);
         break;
	case 3:
         cost = tex2D(mytex3,node%tw,node/tw);
         break;
      }



        row = node/tw + dy[j];
        col = node%tw + dx[j];
              DEBUG("Pre-candidate in R %d (d=%f)\n",row*tw + col,d[node]+cost,dR [4*RLoc[row*tw + col]+j]);
        if( (row>=0) && (row < th) && (col >= 0) && (col < tw) ){ 
            DEBUG("Candidate in R %d (d=%f)\n",row*tw + col,d[node]+cost,dR [4*RLoc[row*tw + col]+j]);
          if((cost>DELTA)&&( d[node]+cost < d[row*tw + col])){
              R [4*RLoc[row*tw + col]+j] = row*tw + col;
              dR[4*RLoc[row*tw + col]+j] = d[node]+cost;
              DEBUG("New node in R %d (d=%f,e=%f) in pos %d\n",R [4*RLoc[row*tw + col]+j],dR [4*RLoc[row*tw + col]+j],cost,4*RLoc[row*tw + col]+j);
              vBucketMap[node]=-1;
          }
        }
      }
    }
  }
  __syncthreads();

  //gathering data to find the minimum cost way to get to node n
  //TODO: OPTIMIZE IN SUCH A WAY IT WON'T BE NEEDED TO GO THROUGH THE 4 EDGES, since they store the same value
  for(int k=0;  (num_threads*k + tid) < 16*BiCount;k++){
    f1 = fminf( dR[4*(num_threads*k + tid)  ], dR[4*(num_threads*k + tid)+1] );
    f2 = fminf( dR[4*(num_threads*k + tid)+2], dR[4*(num_threads*k + tid)+3] );
    fmin = fminf(f1,f2);
    dR[4*(num_threads*k + tid)  ]=fmin;
    dR[4*(num_threads*k + tid)+1]=fmin;
    dR[4*(num_threads*k + tid)+2]=fmin;
    dR[4*(num_threads*k + tid)+3]=fmin;
  }
  __syncthreads();
  
}



__global__ void
emptyKernel(){
}
// B is the bucket i vector
// RLoc[n] stores the position of node n in R (so that if more than one attempt to update
// the distance to node n is made at the same time, it can be shifted to 0,1,2 or 3 in the position of R)


#endif // #ifndef _MEMORY_KERNEL_H_
